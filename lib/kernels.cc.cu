#include "hip/hip_runtime.h"
// This file contains the GPU implementation of our op. It's a pretty typical CUDA kernel
// and I make no promises about the quality of the code or the choices made therein, but
// it should get the point accross.

#include "kdknn.h"
#include "kernel_helpers.h"
#include "kernels.h"

namespace kdknn_jax {

namespace {

template <typename T>
__global__ void kdknn_kernel(std::int64_t size, const T *mean_anom, const T *ecc, T *sin_ecc_anom,
                              T *cos_ecc_anom) {
  for (std::int64_t idx = blockIdx.x * blockDim.x + threadIdx.x; idx < size;
       idx += blockDim.x * gridDim.x) {
    compute_eccentric_anomaly<T>(mean_anom[idx], ecc[idx], sin_ecc_anom + idx, cos_ecc_anom + idx);
  }
}

void ThrowIfError(hipError_t error) {
  if (error != hipSuccess) {
    throw std::runtime_error(hipGetErrorString(error));
  }
}

template <typename T>
inline void apply_kdknn(hipStream_t stream, void **buffers, const char *opaque,
                         std::size_t opaque_len) {
  const KdknnDescriptor &d = *UnpackDescriptor<KdknnDescriptor>(opaque, opaque_len);
  const std::int64_t size = d.size;

  const T *mean_anom = reinterpret_cast<const T *>(buffers[0]);
  const T *ecc = reinterpret_cast<const T *>(buffers[1]);
  T *sin_ecc_anom = reinterpret_cast<T *>(buffers[2]);
  T *cos_ecc_anom = reinterpret_cast<T *>(buffers[3]);

  const int block_dim = 128;
  const int grid_dim = std::min<int>(1024, (size + block_dim - 1) / block_dim);
  kdknn_kernel<T>
      <<<grid_dim, block_dim, 0, stream>>>(size, mean_anom, ecc, sin_ecc_anom, cos_ecc_anom);

  ThrowIfError(hipGetLastError());
}

}  // namespace

void gpu_kdknn_f32(hipStream_t stream, void **buffers, const char *opaque,
                    std::size_t opaque_len) {
  apply_kdknn<float>(stream, buffers, opaque, opaque_len);
}

void gpu_kdknn_f64(hipStream_t stream, void **buffers, const char *opaque,
                    std::size_t opaque_len) {
  apply_kdknn<double>(stream, buffers, opaque, opaque_len);
}

}  // namespace kdknn_jax
